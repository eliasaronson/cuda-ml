#include "hip/hip_runtime.h"
#include "online_regression.h"
#include "utils.cuh"
#include <hip/hip_cooperative_groups.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <stdexcept>

namespace {
// XXᵀ is square so the leading dimension and the minimum dimension is the same
__global__ void add_ridge(double *A, double ridge, int lead_dim) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < lead_dim) {
        A[i * lead_dim + i] += ridge;
    }
}

void transpose(hipblasHandle_t &cublas_handle, double *A, double *A_clone,
               double *C, size_t m, size_t n) {
    double alpha = 1, beta = 0;
    cudaErrorCheck(hipMemcpy(A_clone, A, sizeof(double) * m * n,
                              hipMemcpyDeviceToDevice));

    hipblasDgeam(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, &alpha, A_clone,
                n, &beta, A_clone, m, C, m);
}

namespace cg = cooperative_groups;

template <int tile_sz>
__device__ double reduce_sum_tile_shfl(cg::thread_block_tile<tile_sz> g,
                                       double val) {
    // Each iteration halves the number of active threads
    // Each thread adds its partial sum[i] to sum[lane-i]
    for (int i = g.size() / 2; i > 0; i /= 2) {
        val += g.shfl_down(val, i);
    }

    // Thread 0 returns the sum
    return val;
}

/* ----- R2 numerator ----- */
// Handles a double vectors as double4 to do simd instructions. TODO: Check if
// these actually exists for doubles
__device__ double thread_sumed_sqrt_diff(const double4 *y_true,
                                         const double4 *y_pred, int n) {
    double sum = 0;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n / 4;
         i += blockDim.x * gridDim.x) {
        double4 diff = y_true[i] - y_pred[i];
        diff = diff * diff;
        sum += diff.x + diff.y + diff.z + diff.w;
    }
    return sum;
}

template <int tile_sz>
__global__ void r2_numerator(double *sum, const double *y_true,
                             const double *y_pred, int n) {
    // Allow fewer threads than elements
    double thread_sum =
        thread_sumed_sqrt_diff((double4 *)y_true, (double4 *)y_pred, n);

    auto tile = cg::tiled_partition<tile_sz>(cg::this_thread_block());
    double tile_sum = reduce_sum_tile_shfl<tile_sz>(tile, thread_sum);

    if (tile.thread_rank() == 0) {
        atomicAdd(sum, tile_sum);
    }
}

/* ----- R2 denominator ----- */
// Might be worth to transpose y_true to avoid the modulus operation and for
// better access pattern of y_avg.
__device__ double thread_sumed_sqrt_diff(const double *y_true,
                                         const double *y_avg, int n_features,
                                         int n) {
    double sum = 0;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n;
         i += blockDim.x * gridDim.x) {
        int j = i % n_features;
        double diff = y_true[i] - y_avg[j];
        sum += diff * diff;
    }
    return sum;
}

template <int tile_sz>
__global__ void r2_denominator(double *sum, const double *y_true,
                               const double *y_avg, int n_features, int n) {
    // Allow fewer threads than elements
    double thread_sum = thread_sumed_sqrt_diff(y_true, y_avg, n_features, n);

    auto tile = cg::tiled_partition<tile_sz>(cg::this_thread_block());
    double tile_sum = reduce_sum_tile_shfl<tile_sz>(tile, thread_sum);

    if (tile.thread_rank() == 0) {
        atomicAdd(sum, tile_sum);
    }
}
} // namespace

namespace ml {
online_regression::online_regression(double ridge, int max_iters,
                                     double tolerance)
    : ridge(ridge), max_iters(max_iters), tolerance(tolerance) {
    cublas_handle = new hipblasHandle_t;
    cublasErrorCheck(hipblasCreate(cublas_handle));
    int device;
    hipGetDevice(&device);

    struct hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);
    max_threads_per_block = props.maxThreadsPerBlock;
}

online_regression::~online_regression() {
    clear();
    hipblasDestroy(*cublas_handle);
    delete cublas_handle;
}

std::vector<double> online_regression::predict(const std::vector<double> &X,
                                               size_t X_features,
                                               size_t Y_features) {
    size_t n_samples = X.size() / X_features;
    std::vector<double> Y(Y_features * n_samples);

    double *d_X, *d_Y;

    cudaErrorCheck(hipMalloc(&d_X, sizeof(double) * X.size()));
    cudaErrorCheck(hipMalloc(&d_Y, sizeof(double) * Y.size()));
    cudaErrorCheck(hipMemcpy(d_X, X.data(), sizeof(double) * X.size(),
                              hipMemcpyHostToDevice));

    printf("X_feat; %u, Y_feat: %u, n_samples: %u\n", X_features, Y_features,
           n_samples);
    predict(d_X, d_Y, X_features, Y_features, n_samples);

    cudaErrorCheck(hipMemcpy(Y.data(), d_Y, sizeof(double) * Y.size(),
                              hipMemcpyDeviceToHost));

    printm("Y pred", d_Y, Y_features, n_samples);
    cudaErrorCheck(hipFree(d_X));
    cudaErrorCheck(hipFree(d_Y));

    return Y;
}

// Y = W * X. Y is used as an output and needs to allocated from the outside
double *online_regression::predict(double *X, double *Y, size_t X_m, size_t Y_m,
                                   size_t XY_n) {
    double alpha = 1, beta = 0;

    printm("X test", X, X_m, XY_n);

    // Y = W * X
    cublasErrorCheck(hipblasDgemm(*cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Y_m,
                                 XY_n, X_m, &alpha, W, Y_m, X, X_m, &beta, Y,
                                 Y_m));

    return Y;
}

void online_regression::partial_fit(const std::vector<double> &X,
                                    size_t X_features,
                                    const std::vector<double> &Y,
                                    size_t Y_features) {
    if (X.size() / X_features != Y.size() / Y_features) {
        printf("Error! X and Y have different number of samples.\n");
        return;
    }

    double *d_X, *d_Y;
    hipMalloc(&d_X, sizeof(double) * X.size());
    hipMalloc(&d_Y, sizeof(double) * Y.size());

    hipMemcpy(d_X, X.data(),
               X.size() * sizeof(std::decay_t<decltype(X)>::value_type),
               hipMemcpyHostToDevice);
    hipMemcpy(d_Y, Y.data(),
               Y.size() * sizeof(std::decay_t<decltype(Y)>::value_type),
               hipMemcpyHostToDevice);

    partial_fit(d_X, d_Y, X_features, Y_features, X.size() / X_features);

    cudaErrorCheck(hipFree(d_X));
    cudaErrorCheck(hipFree(d_Y));
}

/* Our goal is to solve W in Y = WX, where X, Y and W are matrices.
 * Y N y features by N samples, W is N y features by N x features and X is
 * N x features by N samples.
 * If we get continuous online data or we * have to much data to load into
 * memory we can compress this data, by the method from this paper:
 * https://www.ai.rug.nl/minds/uploads/PracticalESN.pdf
 *
 * If we multiply both sides of the equation with the transpose of X, the
 * sample dimensions disappears. We now have the following equation instead:
 * YXᵀ = WXXᵀ
 *
 * Each time we run the partial fit we can you add the new XXᵀ and YXᵀ
 * matrices, to the old one. This does run into precision problems for large
 * data sets however. The paper suggests using hierarchical multistage
 * summation or Kahan summation for better precision, which could be added in
 * the future.
 */
void online_regression::partial_fit(double *X, double *Y, size_t X_m,
                                    size_t Y_m, size_t XY_n) {
    double alpha = 1, beta = 0;

    // printm("X", X, X_m, XY_n);
    // printm("Y", Y, Y_m, XY_n);

    // Allocate temporary and accumulation matrices. Don't reallocate these each
    // time for performance reasons.
    if (num_x_features == 0) {
        hipMalloc(&YXt, sizeof(double) * Y_m * X_m);
        hipMalloc(&XXt, sizeof(double) * X_m * X_m);

        hipMalloc(&YXt_partial, sizeof(double) * Y_m * X_m);
        hipMemset(YXt_partial, 0, sizeof(double) * Y_m * X_m);

        hipMalloc(&XXt_partial, sizeof(double) * X_m * X_m);
        hipMemset(XXt_partial, 0, sizeof(double) * X_m * X_m);

        num_x_features = X_m;
        num_y_features = Y_m;
    }

    if (X_m != num_x_features || Y_m != num_y_features) {
        printf("Warning! The number of features does not match previous "
               "partial_fit dimensions.\nPrevious X features: %u, Y features: "
               "%u\nNew X features: %u, Y features: %u.\nPlease run clear() if "
               "you "
               "want to train a new model.\n",
               num_x_features, num_y_features, X_m, Y_m);
        return;
    }

    // Y * Xᵀ
    cublasErrorCheck(hipblasDgemm(*cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, Y_m,
                                 X_m, XY_n, &alpha, Y, Y_m, X, X_m, &beta, YXt,
                                 Y_m));
    printm("YXt", YXt, Y_m, X_m);

    // X * Xᵀ
    cublasErrorCheck(hipblasDgemm(*cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, X_m,
                                 X_m, XY_n, &alpha, X, X_m, X, X_m, &beta, XXt,
                                 X_m));

    printm("XXt", XXt, X_m, X_m);

    // Accumulate YXᵀ and XXᵀ for further partial fits or to fully solve
    cublasErrorCheck(hipblasDgeam(*cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Y_m,
                                 X_m, &alpha, YXt_partial, Y_m, &alpha, YXt,
                                 Y_m, YXt_partial, Y_m));

    printm("YXt_partial", YXt_partial, Y_m, X_m);

    cublasErrorCheck(hipblasDgeam(*cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, X_m,
                                 X_m, &alpha, XXt_partial, X_m, &alpha, XXt,
                                 X_m, XXt_partial, X_m));
    printm("XXt_partial", XXt_partial, X_m, X_m);
}

void online_regression::fit(const std::vector<double> &X, size_t X_features,
                            const std::vector<double> &Y, size_t Y_features) {
    partial_fit(X, X_features, Y, Y_features);
    fit(nullptr, nullptr, X_features, Y_features, X.size() / X_features);
}

/* Cusolver's gels solves for X in AX = B.
 * We want to solve for W in YXᵀ = WXXᵀ.
 *
 * To be able to perform the partial fits we need to have X last, but that also
 * means we need to transpose both sides to fit the format of the solver.
 * To do this  we just need to transpose both sides.
 * (YXᵀ)ᵀ = (WXXᵀ)ᵀ
 * (YXᵀ)ᵀ = (XXᵀ)ᵀWᵀ
 * (YXᵀ)ᵀ = XXᵀWᵀ
 *
 * This gives:
 * A = XXᵀ
 * B = (YXᵀ)ᵀ
 * X = W
 * */
void online_regression::fit(double *X, double *Y, size_t X_m, size_t Y_m,
                            size_t XY_n) {
    if (Y != nullptr && Y != nullptr) {
        partial_fit(X, Y, X_m, Y_m, XY_n);
    }

    printm("A", XXt_partial, X_m, X_m);

    if (ridge != 0) {
        printf("ridge: %f\n", ridge);
        int block_size;
        int min_grid_size;
        int grid_size;

        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                                           add_ridge, 0, 0);

        grid_size = (X_m + block_size - 1) / block_size;
        printm("Pred ridge A", XXt_partial, X_m, X_m);
        add_ridge<<<grid_size, block_size>>>(XXt_partial, ridge, X_m);
        printm("Post ridge A", XXt_partial, X_m, X_m);
    }

    // B is now (YXᵀ)ᵀ. tmp is used for temporary storage in transposes.
    double *B, *tmp;
    hipMalloc(&B, sizeof(double) * X_m * Y_m);
    hipMemset(B, 0, sizeof(double) * X_m * Y_m);

    hipMalloc(&tmp, sizeof(double) * X_m * Y_m);

    transpose(*cublas_handle, YXt_partial, tmp, B, Y_m, X_m);

    printm("B", B, X_m, Y_m);

    /* Allocate data for settings and work */
    int niter = 0;
    size_t work_size = 0;

    double *Wt = nullptr;
    double *work = nullptr;
    int *info = nullptr;

    hipsolverHandle_t cusolver_handle = nullptr;
    cusolverErrorCheck(hipsolverDnCreate(&cusolver_handle));

    cudaErrorCheck(hipMalloc(&Wt, sizeof(double) * X_m * Y_m));
    cudaErrorCheck(hipMalloc(&info, sizeof(int)));
    cudaErrorCheck(hipMalloc(&W, sizeof(double) * Y_m * X_m));

    cusolverDnIRSParams_t gels_irs_params;
    cusolverDnIRSParamsCreate(&gels_irs_params);

    cusolverDnIRSInfos_t gels_irs_infos;
    cusolverDnIRSInfosCreate(&gels_irs_infos);
    cusolverErrorCheck(cusolverDnIRSInfosRequestResidual(gels_irs_infos));

    /* Set all settings */
    // Solver precisions
    cusolverErrorCheck(cusolverDnIRSParamsSetSolverPrecisions(
        gels_irs_params, CUSOLVER_R_64F, CUSOLVER_R_64F));

    // Refinement solver.
    if (Y_m == 1) {
        // Generalized Minimal Residual is more accurate, but can only solve for
        // one right hand side. TODO: Maybe add option to set
        // CUSOLVER_IRS_REFINE_CLASSICAL_GMRES or
        // CUSOLVER_IRS_REFINE_GMRES_GMRES.
        cusolverErrorCheck(cusolverDnIRSParamsSetRefinementSolver(
            gels_irs_params, CUSOLVER_IRS_REFINE_GMRES));
    } else {
        cusolverErrorCheck(cusolverDnIRSParamsSetRefinementSolver(
            gels_irs_params, CUSOLVER_IRS_REFINE_CLASSICAL));
    }
    cusolverErrorCheck(
        cusolverDnIRSParamsSetMaxIters(gels_irs_params, max_iters));
    cusolverErrorCheck(cusolverDnIRSParamsSetTol(gels_irs_params, tolerance));

    // Find the work buffer size from the parameters and allocate it
    cusolverErrorCheck(cusolverDnIRSXgels_bufferSize(
        cusolver_handle, gels_irs_params, X_m, X_m, Y_m, &work_size));
    cudaErrorCheck(hipMalloc(&work, work_size));

    /* Run the solver */
    cusolverErrorCheck(cusolverDnIRSXgels(
        cusolver_handle, gels_irs_params, gels_irs_infos, X_m, X_m, Y_m,
        XXt_partial, X_m, B, X_m, Wt, X_m, work, work_size, &niter, info));

    printm("Wt", Wt, X_m, Y_m);

    printf("solver iterations: %i\n", niter);

    // No need to transpose if W is a vector
    if (X_m == 1 || Y_m == 1) {
        cudaErrorCheck(hipMemcpy(W, Wt, sizeof(double) * Y_m * X_m,
                                  hipMemcpyDeviceToDevice));
    } else {
        transpose(*cublas_handle, Wt, tmp, W, X_m, Y_m);
    }
    printm("W", W, Y_m, X_m);

#define CHECK_AND_FREE(PTR, FREE)                                              \
    if (PTR != nullptr) {                                                      \
        FREE(PTR);                                                             \
    }                                                                          \
    PTR = nullptr;

    // Clean up
    CHECK_AND_FREE(tmp, hipFree)
    CHECK_AND_FREE(Wt, hipFree)
    CHECK_AND_FREE(info, hipFree)
    CHECK_AND_FREE(work, hipFree)
    CHECK_AND_FREE(B, hipFree)
    CHECK_AND_FREE(cusolver_handle, hipsolverDnDestroy)

    cusolverErrorCheck(cusolverDnIRSParamsDestroy(gels_irs_params));
    cusolverErrorCheck(cusolverDnIRSInfosDestroy(gels_irs_infos));
}

void online_regression::clear() {
    CHECK_AND_FREE(YXt, hipFree)
    CHECK_AND_FREE(XXt, hipFree)

    CHECK_AND_FREE(YXt_partial, hipFree)
    CHECK_AND_FREE(XXt_partial, hipFree)

    CHECK_AND_FREE(W, hipFree)

    num_x_features = num_y_features = 0;
}

#undef CHECK_AND_FREE

double online_regression::score(const std::vector<double> &X, size_t X_features,
                                const std::vector<double> &Y,
                                size_t Y_features) {
    size_t n_samples = X.size() / X_features;
    size_t pad = num_extra_to_pad(Y.size());

    double *d_X, *d_Y;

    cudaErrorCheck(hipMalloc(&d_X, sizeof(double) * X.size()));
    cudaErrorCheck(hipMalloc(&d_Y, sizeof(double) * (Y.size() + pad)));

    cudaErrorCheck(hipMemcpy(d_X, X.data(), sizeof(double) * X.size(),
                              hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(d_Y, Y.data(), sizeof(double) * Y.size(),
                              hipMemcpyHostToDevice));

    if (pad > 0) {
        cudaErrorCheck(hipMemset(d_Y + Y.size(), 0, pad * sizeof(double)));
    }

    printf("X_feat; %u, Y_feat: %u, n_samples: %u\n", X_features, Y_features,
           n_samples);
    double res = score(d_X, d_Y, X_features, Y_features, n_samples, true);

    cudaErrorCheck(hipFree(d_X));
    cudaErrorCheck(hipFree(d_Y));

    return res;
}

// R2 score. Y should be padded to be divisible by 4 or it will need to be
// reallocated (this is probably not the best solution).
double online_regression::score(double *X, double *Y, size_t X_m, size_t Y_m,
                                size_t XY_n, bool padded) {
    if (XY_n < 2) {
        printf("Warning! R2 requires at least two samples.\n");
    }

    double *Y_padded, *Y_pred, *numerator, *denominator, *Y_average, *Y_div;

    size_t pad = num_extra_to_pad(Y_m * XY_n);
    size_t y_sz = Y_m * XY_n + pad;
    cudaErrorCheck(hipMalloc(&Y_pred, sizeof(double) * y_sz));
    cudaErrorCheck(hipMalloc(&numerator, sizeof(double)));
    cudaErrorCheck(hipMalloc(&denominator, sizeof(double)));
    cudaErrorCheck(hipMalloc(&Y_div, sizeof(double) * XY_n));
    cudaErrorCheck(hipMalloc(&Y_average, sizeof(double) * Y_m));

    // Probably not worth it, unless already padded from the outside
    if (pad > 0) {
        cudaErrorCheck(
            hipMemset(Y_pred + Y_m * XY_n, 0, pad * sizeof(double)));
    }
    if (!padded && pad > 0) {
        cudaErrorCheck(hipMalloc(&Y_padded, sizeof(double) * y_sz));
        cudaErrorCheck(hipMemcpy(Y_padded, Y, sizeof(double) * Y_m * XY_n,
                                  hipMemcpyDeviceToDevice));
        cudaErrorCheck(
            hipMemset(Y_padded + Y_m * XY_n, 0, pad * sizeof(double)));
    } else {
        Y_padded = Y;
    }

    // Get prediction
    predict(X, Y_pred, X_m, Y_m, XY_n);
    printm("Y", Y, Y_m, XY_n);
    printm("Y_pred", Y_pred, Y_m, XY_n);

    // We should not be register limited, so should not be a problem to max out
    // the blocks when we have large matrices. Might be more efficient to use
    // more SMs with small matrices however.
    dim3 block_dim(max_threads_per_block);
    dim3 grid_dim;
    grid_dim.x = (XY_n + block_dim.x - 1) / block_dim.x;
    fill<double><<<grid_dim, block_dim>>>(Y_div, 1. / XY_n, XY_n);

    grid_dim.x = (y_sz + block_dim.x - 1) / block_dim.x;
    r2_numerator<32>
        <<<grid_dim, block_dim>>>(numerator, Y_padded, Y_pred, y_sz);
    printm("numerator", numerator, 1, 1);

    double alpha = 1., beta = 0.;
    cublasErrorCheck(hipblasDgemm(*cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Y_m,
                                 1, XY_n, &alpha, Y, Y_m, Y_div, XY_n, &beta,
                                 Y_average, Y_m));
    printm("avg", Y_average, Y_m, 1);

    r2_denominator<32>
        <<<grid_dim, block_dim>>>(denominator, Y, Y_average, Y_m, Y_m * XY_n);
    printm("denominator", denominator, 1, 1);

    double h_denominator, h_numerator;
    hipMemcpy(&h_numerator, numerator, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&h_denominator, denominator, sizeof(double),
               hipMemcpyDeviceToHost);

    cudaErrorCheck(hipFree(Y_pred));
    cudaErrorCheck(hipFree(numerator));
    cudaErrorCheck(hipFree(denominator));
    cudaErrorCheck(hipFree(Y_div));
    cudaErrorCheck(hipFree(Y_average));
    if (!padded && pad != 0) {
        hipFree(Y_padded);
    }

    return 1. - h_numerator / h_denominator;
}
} // namespace ml
