#include "hip/hip_runtime.h"
#include "online_regression.h"
#include <iostream> // Only needed for debugging

namespace {
void printm(std::string label, double *A, size_t m, size_t n) {
  std::cout << label << " " << m << "x" << n << ":\n";
  double h_A[m * n];
  cudaErrorCheck(
      hipMemcpy(h_A, A, m * n * sizeof(double), hipMemcpyDeviceToHost));

  for (int i = 0; i < m; ++i) {
    for (int j = 0; j < n; ++j) {
      printf("%f", h_A[i + j * m]);
      if (j != n - 1) {
        printf(", ");
      }
    }
    printf("\n");
  }
  printf("\n");
}

// XXᵀ is square so the leading dimension and the minimum dimension is the same
__global__ void add_ridge(double *A, double ridge, int lead_dim) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < lead_dim) {
    A[i * lead_dim + i] += ridge;
  }
}

void transpose(hipblasHandle_t &cublas_handle, double *A, double *A_clone,
               double *C, size_t m, size_t n) {
  double alpha = 1, beta = 0;
  cudaErrorCheck(
      hipMemcpy(A_clone, A, sizeof(double) * m * n, hipMemcpyDeviceToDevice));

  hipblasDgeam(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, &alpha, A_clone, n,
              &beta, A_clone, m, C, m);
}
} // namespace

online_regression::online_regression(double ridge, int max_iters,
                                     double tolerance)
    : ridge(ridge), max_iters(max_iters), tolerance(tolerance) {
  cublasErrorCheck(hipblasCreate(&cublas_handle));
}

online_regression::~online_regression() {
  clear();
  hipblasDestroy(cublas_handle);
}

std::vector<double> online_regression::predict(std::vector<double> X,
                                               size_t X_features,
                                               size_t Y_features) {
  size_t n_samples = X.size() / X_features;
  std::vector<double> Y(Y_features * n_samples);

  double *d_X, *d_Y;

  cudaErrorCheck(hipMalloc(&d_X, sizeof(double) * X.size()));
  cudaErrorCheck(hipMalloc(&d_Y, sizeof(double) * Y.size()));
  cudaErrorCheck(hipMemcpy(d_X, X.data(), sizeof(double) * X.size(),
                            hipMemcpyHostToDevice));

  printf("X_feat; %u, Y_feat: %u, n_samples: %u\n", X_features, Y_features,
         n_samples);
  predict(d_X, d_Y, X_features, Y_features, n_samples);

  cudaErrorCheck(hipMemcpy(Y.data(), d_Y, sizeof(double) * Y.size(),
                            hipMemcpyDeviceToHost));

  printm("Y pred", d_Y, Y_features, n_samples);
  cudaErrorCheck(hipFree(d_X));
  cudaErrorCheck(hipFree(d_Y));

  return Y;
}

// Y = W * X. Y is used as an output and needs to allocated from the outside
double *online_regression::predict(double *X, double *Y, size_t X_m, size_t Y_m,
                                   size_t XY_n) {
  double alpha = 1, beta = 0;

  printm("X test", X, X_m, XY_n);

  // Y = W * X
  cublasErrorCheck(hipblasDgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Y_m,
                               XY_n, X_m, &alpha, W, Y_m, X, X_m, &beta, Y,
                               Y_m));

  return Y;
}

void online_regression::partial_fit(std::vector<double> X, size_t X_features,
                                    std::vector<double> Y, size_t Y_features) {
  if (X.size() / X_features != Y.size() / Y_features) {
    printf("Error! X and Y have different number of samples.\n");
    return;
  }

  double *d_X, *d_Y;
  hipMalloc(&d_X, sizeof(double) * X.size());
  hipMalloc(&d_Y, sizeof(double) * Y.size());

  hipMemcpy(d_X, X.data(), X.size() * sizeof(decltype(X)::value_type),
             hipMemcpyHostToDevice);
  hipMemcpy(d_Y, Y.data(), Y.size() * sizeof(decltype(Y)::value_type),
             hipMemcpyHostToDevice);

  partial_fit(d_X, d_Y, X_features, Y_features, X.size() / X_features);

  cudaErrorCheck(hipFree(d_X));
  cudaErrorCheck(hipFree(d_Y));
}

/* Our goal is to solve W in Y = WX, where X, Y and W are matrices.
 * Y N y features by N samples, W is N y features by N x features and X is
 * N x features by N samples.
 * If we get continuous online data or we * have to much data to load into
 * memory we can compress this data, by the method from this paper:
 * https://www.ai.rug.nl/minds/uploads/PracticalESN.pdf
 *
 * If we multiply both sides of the equation with the transpose of X, the
 * sample dimensions disappears. We now have the following equation instead:
 * YXᵀ = WXXᵀ
 *
 * Each time we run the partial fit we can you add the new XXᵀ and YXᵀ
 * matrices, to the old one. This does run into precision problems for large
 * data sets however. The paper suggests using hierarchical multistage
 * summation or Kahan summation for better precision, which could be added in
 * the future.
 */
void online_regression::partial_fit(double *X, double *Y, size_t X_m,
                                    size_t Y_m, size_t XY_n) {
  double alpha = 1, beta = 0;

  // printm("X", X, X_m, XY_n);
  // printm("Y", Y, Y_m, XY_n);

  // Allocate temporary and accumulation matrices. Don't reallocate these each
  // time for performance reasons.
  if (num_x_features == 0) {
    hipMalloc(&YXt, sizeof(double) * Y_m * X_m);
    hipMalloc(&XXt, sizeof(double) * X_m * X_m);

    hipMalloc(&YXt_partial, sizeof(double) * Y_m * X_m);
    hipMemset(YXt_partial, 0, sizeof(double) * Y_m * X_m);

    hipMalloc(&XXt_partial, sizeof(double) * X_m * X_m);
    hipMemset(XXt_partial, 0, sizeof(double) * X_m * X_m);

    num_x_features = X_m;
    num_y_features = Y_m;
  }

  if (X_m != num_x_features || Y_m != num_y_features) {
    printf("Warning! The number of features does not match previous "
           "partial_fit dimensions.\nPrevious X features: %u, Y features: "
           "%u\nNew X features: %u, Y features: %u.\nPlease run clear() if you "
           "want to train a new model.\n",
           num_x_features, num_y_features, X_m, Y_m);
    return;
  }

  // Y * Xᵀ
  cublasErrorCheck(hipblasDgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, Y_m,
                               X_m, XY_n, &alpha, Y, Y_m, X, X_m, &beta, YXt,
                               Y_m));
  printm("YXt", YXt, Y_m, X_m);

  // X * Xᵀ
  cublasErrorCheck(hipblasDgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, X_m,
                               X_m, XY_n, &alpha, X, X_m, X, X_m, &beta, XXt,
                               X_m));

  printm("XXt", XXt, X_m, X_m);

  // Accumulate YXᵀ and XXᵀ for further partial fits or to fully solve
  cublasErrorCheck(hipblasDgeam(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Y_m,
                               X_m, &alpha, YXt_partial, Y_m, &alpha, YXt, Y_m,
                               YXt_partial, Y_m));

  printm("YXt_partial", YXt_partial, Y_m, X_m);

  cublasErrorCheck(hipblasDgeam(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, X_m,
                               X_m, &alpha, XXt_partial, X_m, &alpha, XXt, X_m,
                               XXt_partial, X_m));
  printm("XXt_partial", XXt_partial, X_m, X_m);
}

void online_regression::fit(std::vector<double> X, size_t X_features,
                            std::vector<double> Y, size_t Y_features) {
  partial_fit(X, X_features, Y, Y_features);
  fit(nullptr, nullptr, X_features, Y_features, X.size() / X_features);
}

/* Cusolver's gels solves for X in AX = B.
 * We want to solve for W in YXᵀ = WXXᵀ.
 *
 * To be able to perform the partial fits we need to have X last, but that also
 * means we need to transpose both sides to fit the format of the solver.
 * To do this  we just need to transpose both sides.
 * (YXᵀ)ᵀ = (WXXᵀ)ᵀ
 * (YXᵀ)ᵀ = (XXᵀ)ᵀWᵀ
 * (YXᵀ)ᵀ = XXᵀWᵀ
 *
 * This gives:
 * A = XXᵀ
 * B = (YXᵀ)ᵀ
 * X = W
 * */
void online_regression::fit(double *X, double *Y, size_t X_m, size_t Y_m,
                            size_t XY_n) {
  if (Y != nullptr && Y != nullptr) {
    partial_fit(X, Y, X_m, Y_m, XY_n);
  }

  printm("A", XXt_partial, X_m, X_m);

  if (ridge != 0) {
    printf("ridge: %f\n", ridge);
    int block_size;
    int min_grid_size;
    int grid_size;

    hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, add_ridge,
                                       0, 0);

    grid_size = (X_m + block_size - 1) / block_size;
    printm("Pred ridge A", XXt_partial, X_m, X_m);
    add_ridge<<<grid_size, block_size>>>(XXt_partial, ridge, X_m);
    printm("Post ridge A", XXt_partial, X_m, X_m);
  }

  // B is now (YXᵀ)ᵀ. tmp is used for temporary storage in transposes.
  double *B, *tmp;
  hipMalloc(&B, sizeof(double) * X_m * Y_m);
  hipMemset(B, 0, sizeof(double) * X_m * Y_m);

  hipMalloc(&tmp, sizeof(double) * X_m * Y_m);

  transpose(cublas_handle, YXt_partial, tmp, B, Y_m, X_m);

  printm("B", B, X_m, Y_m);

  /* Allocate data for settings and work */
  int niter = 0;
  size_t work_size = 0;

  double *Wt = nullptr;
  double *work = nullptr;
  int *info = nullptr;

  hipsolverHandle_t cusolver_handle = nullptr;
  cusolverErrorCheck(hipsolverDnCreate(&cusolver_handle));

  cudaErrorCheck(hipMalloc(&Wt, sizeof(double) * X_m * Y_m));
  cudaErrorCheck(hipMalloc(&info, sizeof(int)));
  cudaErrorCheck(hipMalloc(&W, sizeof(double) * Y_m * X_m));

  cusolverDnIRSParams_t gels_irs_params;
  cusolverDnIRSParamsCreate(&gels_irs_params);

  cusolverDnIRSInfos_t gels_irs_infos;
  cusolverDnIRSInfosCreate(&gels_irs_infos);
  cusolverErrorCheck(cusolverDnIRSInfosRequestResidual(gels_irs_infos));

  /* Set all settings */
  // Solver precisions
  cusolverErrorCheck(cusolverDnIRSParamsSetSolverPrecisions(
      gels_irs_params, CUSOLVER_R_64F, CUSOLVER_R_64F));

  // Refinement solver.
  if (Y_m == 1) {
    // Generalized Minimal Residual is more accurate, but can only solve for
    // one right hand side. TODO: Maybe add option to set
    // CUSOLVER_IRS_REFINE_CLASSICAL_GMRES or CUSOLVER_IRS_REFINE_GMRES_GMRES.
    cusolverErrorCheck(cusolverDnIRSParamsSetRefinementSolver(
        gels_irs_params, CUSOLVER_IRS_REFINE_GMRES));
  } else {
    cusolverErrorCheck(cusolverDnIRSParamsSetRefinementSolver(
        gels_irs_params, CUSOLVER_IRS_REFINE_CLASSICAL));
  }
  cusolverErrorCheck(
      cusolverDnIRSParamsSetMaxIters(gels_irs_params, max_iters));
  cusolverErrorCheck(cusolverDnIRSParamsSetTol(gels_irs_params, tolerance));

  // Find the work buffer size from the parameters and allocate it
  cusolverErrorCheck(cusolverDnIRSXgels_bufferSize(
      cusolver_handle, gels_irs_params, X_m, X_m, Y_m, &work_size));
  cudaErrorCheck(hipMalloc(&work, work_size));

  /* Run the solver */
  cusolverErrorCheck(cusolverDnIRSXgels(
      cusolver_handle, gels_irs_params, gels_irs_infos, X_m, X_m, Y_m,
      XXt_partial, X_m, B, X_m, Wt, X_m, work, work_size, &niter, info));

  printm("Wt", Wt, X_m, Y_m);

  printf("solver iterations: %i\n", niter);

  // No need to transpose if W is a vector
  if (X_m == 1 || Y_m == 1) {
    cudaErrorCheck(hipMemcpy(W, Wt, sizeof(double) * Y_m * X_m,
                              hipMemcpyDeviceToDevice));
  } else {
    transpose(cublas_handle, Wt, tmp, W, X_m, Y_m);
  }
  printm("W", W, Y_m, X_m);

#define CHECK_AND_FREE(PTR, FREE)                                              \
  if (PTR != nullptr) {                                                        \
    FREE(PTR);                                                                 \
  }

  // Clean up
  CHECK_AND_FREE(tmp, hipFree)
  CHECK_AND_FREE(Wt, hipFree)
  CHECK_AND_FREE(info, hipFree)
  CHECK_AND_FREE(work, hipFree)
  CHECK_AND_FREE(B, hipFree)
  CHECK_AND_FREE(cusolver_handle, hipsolverDnDestroy)

  cusolverErrorCheck(cusolverDnIRSParamsDestroy(gels_irs_params));
  cusolverErrorCheck(cusolverDnIRSInfosDestroy(gels_irs_infos));
}

void online_regression::clear() {
  CHECK_AND_FREE(YXt, hipFree)
  CHECK_AND_FREE(XXt, hipFree)

  CHECK_AND_FREE(YXt_partial, hipFree)
  CHECK_AND_FREE(XXt_partial, hipFree)

  CHECK_AND_FREE(W, hipFree)

  num_x_features = num_y_features = 0;
}

#undef CHECK_AND_FREE

void score(std::vector<double> X, size_t X_features, std::vector<double> Y,
           size_t Y_features) {}

void score(double *X, double *Y, size_t X_m, size_t Y_m, size_t XY_n) {}
